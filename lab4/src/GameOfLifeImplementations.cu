#include "hip/hip_runtime.h"
/*
Author: Ben Hall
Class: ECE6122 (section)
Last Date Modified: 11/08/24
Description: file containing the global cuda functions that will get run on a cuda device for updating the game of life grid
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <GameOfLifeImplementations.hpp>
#include <iostream>
#include <stdio.h>

// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/3_CUDA_Features/newdelete/newdelete.cu
// https://stackoverflow.com/questions/12373940/difference-between-global-and-device-functions

/// @brief function to get the number of neighbors around a coordinate 
/// @param state pointer to the grid itself 
/// @param x the width to check 
/// @param y the height to check
/// @param width width of the grid itself
/// @param height height of the grid
/// @return neighbors that are alive around the desired coordinate
__device__ int countNeighbors(bool *state, std::size_t x, std::size_t y, int width, int height)
{
    int neighbors = 0;
    for (int dy = -1; dy <= 1; ++dy)
    {
        for (int dx = -1; dx <= 1; ++dx)
        {
            if (dx == 0 && dy == 0) continue; 

            int nx = (x + dx + width) % width;
            int ny = (y + dy + height) % height;

            neighbors += state[nx + (ny * width)]; // Add the neighbor's state
        }
    }
    return neighbors;
}

/// @brief function that actually updates the grid. works on the CUDA allocated grids
/// @param current_state the pointer to the current grid
/// @param prev_state pointer to the previous grid
/// @param width width of grid
/// @param height height of grid
__global__ void updateGrid(bool *current_state, bool *prev_state, int width, int height)
{
    int index_x = threadIdx.x;
    int index_y = threadIdx.y;
    
    int stride_x = blockDim.x;
    int stride_y = blockDim.y;

    for (int y = index_y; y < height; y+=stride_y)
    {
        for (int x = index_x; x < width; x+=stride_x)
        {
            int neighbors = countNeighbors(prev_state, (std::size_t)x, (std::size_t)y, width, height);
            int index = x + (y * width);  
            
            if (prev_state[index]) 
            {
                if (neighbors < 2 || neighbors > 3)
                {
                    current_state[index] = false; 
                }
                else
                {
                    current_state[index] = prev_state[index]; 
                }
            }
            else 
            {
                if (neighbors == 3)
                {
                    current_state[index] = true; 
                }
                else
                {
                    current_state[index] = false; 
                }
            }
        }
    }
}
