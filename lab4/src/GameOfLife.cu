#include "hip/hip_runtime.h"
/*
Author: Ben Hall
Class: ECE6122 (section)
Last Date Modified: 11/08/24
Description: file containing the implementation of the GoL class that wraps the CUDA functions implemented
*/

#include <GameOfLife.hpp>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <cmath>

#include <algorithm>    // std::swap

#define DEBUG true
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

GoL::GoL(std::size_t width, std::size_t height, std::size_t n, int cudaMode)
{
    // row by col: height by width
    
    _heightSize = height;
    _widthSize = width;
    _mode = cudaMode;
    _numThreads = n;
    auto b = sqrt(_numThreads);
    dim3 blockSize(b, b);  // block of threads
    dim3 gridSize((_widthSize + blockSize.x - 1) / blockSize.x, (_widthSize + blockSize.y - 1) / blockSize.y);
    _blockSize = blockSize;
    _gridSize = gridSize;

    
    _displayMatrixTest = static_cast<bool *>(malloc(sizeof(bool)*height*width));
    switch(cudaMode)
    {
        case 0:
        {
            _displayMatrix = static_cast<bool *>(malloc(sizeof(bool)*height*width));
            _generateGrid(width, height, _displayMatrix);
            _allocateNormal(&_devicePtrCurrenState, width, height);
            _allocateNormal(&_devicePtrPrevState, width, height);
            _copyGridNormalToDev(_displayMatrix, _devicePtrCurrenState);
            std::cout << "copied grid prev" <<std::endl;
            _copyGridNormalToDev(_displayMatrix, _devicePtrPrevState);
            std::cout << "copied grid curr" <<std::endl;
            break;
        }
        case 1:
        {
            _allocatePinned(&_displayMatrix, &_devicePtrCurrenState, &_devicePtrPrevState, width, height);
            _generateGrid(width, height, _displayMatrix);
            
            _copyGridNormalToDev(_displayMatrix, _devicePtrCurrenState);
            _copyGridNormalToDev(_displayMatrix, _devicePtrPrevState);
            break;
        }
        case 2:
        {
            // turns the prev device ptr into a managed one that we can use as well
            _allocateManagedAndSync(&_displayMatrix, &_devicePtrPrevState, width, height);
            _generateGrid(width, height, _displayMatrix);
            _generateGrid(width, height, _devicePtrPrevState);
        }
        default:
        {
            break;
        }
    }
}

GoL::~GoL()
{
    free(_displayMatrixTest);
    switch(_mode)
    {
        case 0:
        {
            free(_displayMatrix);
            _deallocNormal(_devicePtrCurrenState);
            _deallocNormal(_devicePtrPrevState);
            break;
        }
        case 1:
        {
            _deallocPinned(_displayMatrix);
            _deallocNormal(_devicePtrCurrenState);
            _deallocNormal(_devicePtrPrevState);
            break;
        }
        case 2:
        {
           _deallocNormal(_displayMatrix); 
           _deallocNormal(_devicePtrPrevState);
        }
        default:
        {
            break;
        }
    } 
}

void GoL::_allocateNormal(bool **dev_ptr, std::size_t width, std::size_t height)
{
    checkCuda(hipMalloc((void **)dev_ptr, height * width * sizeof(bool)));
}

void GoL::_allocatePinned(bool **host_ptr, bool **dev_ptr, bool **other_dev_ptr, std::size_t width, std::size_t height)
{
    checkCuda(hipHostMalloc((void**)host_ptr, height * width * sizeof(bool)));
    checkCuda(hipMalloc((void **)dev_ptr, height * width * sizeof(bool)));
    checkCuda(hipMalloc((void **)other_dev_ptr, height * width * sizeof(bool)));
}

void GoL::_allocateManagedAndSync(bool **host_ptr, bool **dev_ptr, std::size_t width, std::size_t height)
{
    checkCuda(hipMallocManaged((void**)host_ptr, height * width * sizeof(bool)));
    checkCuda(hipMallocManaged((void **)dev_ptr, height * width * sizeof(bool)));
    hipDeviceSynchronize();
}

void GoL::_deallocNormal(bool *dev_ptr)
{
    hipFree(dev_ptr);
}

void GoL::_deallocPinned(bool *host_ptr)
{
    hipHostFree(host_ptr);
}

void GoL::_copyGridNormal(bool *dev_ptr, bool *host_ptr) {
    if(!dev_ptr || !host_ptr)
    {
        if(!dev_ptr)
        {
            std::cout << "WARNING: dev ptr null" <<std::endl;
        }
        if(!host_ptr)
        {
            std::cout << "WARNING: host ptr null" <<std::endl;
        }
        return;
    }
    // Copy the device pointer array to host first
    hipMemcpy(host_ptr, dev_ptr, sizeof(bool) * _heightSize * _widthSize, hipMemcpyDeviceToHost);
}

void GoL::_copyGridPinned(bool *dev_ptr, bool *host_ptr) {
    if(!dev_ptr || !host_ptr)
    {
        if(!dev_ptr)
        {
            std::cout << "WARNING: dev ptr null" <<std::endl;
        }
        if(!host_ptr)
        {
            std::cout << "WARNING: host ptr null" <<std::endl;
        }
        return;
    }
    // Copy the device pointer array to host first
    hipDeviceSynchronize();
    hipMemcpy(host_ptr, dev_ptr, sizeof(bool) * _heightSize * _widthSize, hipMemcpyDeviceToHost);
}

void GoL::_copyGridNormalToDev(bool *host_ptr, bool *dev_ptr) {
    if(!dev_ptr || !host_ptr)
    {
        if(!dev_ptr)
        {

            std::cout << "WARNING: dev ptr null" <<std::endl;
        }
        if(!host_ptr)
        {
            std::cout << "WARNING: host ptr null" <<std::endl;
        }
        return;
    }
    // Copy the device pointer array to host first
    hipMemcpy(dev_ptr, host_ptr, sizeof(bool) * _heightSize * _widthSize, hipMemcpyHostToDevice);
}

bool GoL::_generateGrid(std::size_t width, std::size_t height, bool *grid)
{
    if(!grid)
    {
        return false;
    }

    std::srand(static_cast<unsigned>(std::time(nullptr)));
    for (std::size_t x = 0; x < width; ++x)
    {
        for (std::size_t y = 0; y < height; ++y)
        {
            grid[x + (y * width)] = (std::rand() % 2 == 0); // Randomly seed each pixel
        }
    }
    return true;
}

bool * GoL::getGrid()
{
    // _updateGridTest(_displayMatrixTest, _displayMatrix, _widthSize, _heightSize);
    // std::swap(_displayMatrix, _displayMatrixTest);
    switch(_mode)
    {
        case 0:
        case 1:
        {
            updateGrid<<<_gridSize, _blockSize>>>( _devicePtrCurrenState, _devicePtrPrevState, _widthSize, _heightSize);
            _copyGridNormal(_devicePtrCurrenState, _displayMatrix);
            std::swap(_devicePtrCurrenState, _devicePtrPrevState);
            break;
        }
        case 2:
        {
            updateGrid<<<_gridSize, _blockSize>>>(_displayMatrix, _devicePtrPrevState, _widthSize, _heightSize);
            hipDeviceSynchronize();
            std::swap(_displayMatrix, _devicePtrPrevState);
            break;
        }
        default:
        {
            break;
        }
    }
    
    return _displayMatrix;
}

// test functions for 1d to 2d un-flattened test
int GoL::_countNeighborsTest(bool *state, int x, int y, std::size_t width, std::size_t height)
{
    int neighbors = 0;
    for (int dy = -1; dy <= 1; ++dy)
    {
        for (int dx = -1; dx <= 1; ++dx)
        {
            if (dx == 0 && dy == 0) continue; 

            int nx = (x + dx + width) % width; 
            int ny = (y + dy + height) % height; 

            neighbors += state[nx + (ny * width)]; 
        }
    }
    return neighbors;
}

void GoL::_updateGridTest(bool *current_state, bool *prev_state, std::size_t width, std::size_t height)
{
    for (int y = 0; y < height; ++y)
    {
        for (int x = 0; x < width; ++x)
        {
            int neighbors = _countNeighborsTest(prev_state, x, y, width, height);
            int index = x + (y * width);
            
            if (prev_state[index])
            {
                if (neighbors < 2 || neighbors > 3)
                {
                    current_state[index] = false;
                }
                else
                {
                    current_state[index] = prev_state[index];
                }
            }
            else
            {
                if (neighbors == 3)
                {
                    current_state[index] = true;
                }
                else
                {
                    current_state[index] = false;
                }
            }
        }
    }
}